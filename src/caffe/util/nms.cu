#include "hip/hip_runtime.h"
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/remove.h>

#include "caffe/util/nms.hpp"
#include "caffe/util/gpu_util.cuh"

namespace caffe {


struct thrust_is_zero
{
  __device__
  bool operator()(const int x) const {
    return ( !x ); }
};

template<typename Dtype>
__global__ void nms_kernel(const int boxes_num, const int* indexes,
                           const Dtype *boxes, unsigned long long *dev_mask,
                           const Dtype nms_overlap_thresh ) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;
  // if (row_start > col_start) return;

  const int row_size =
          min(boxes_num - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
          min(boxes_num - col_start * threadsPerBlock, threadsPerBlock);
  __shared__ Dtype block_boxes[threadsPerBlock*4];
  if (threadIdx.x < col_size) {
    int index = indexes[(threadsPerBlock * col_start + threadIdx.x)];
    block_boxes[threadIdx.x * 4 + 0] = boxes[index * 4 + 0];
    block_boxes[threadIdx.x * 4 + 1] = boxes[index * 4 + 1];
    block_boxes[threadIdx.x * 4 + 2] = boxes[index * 4 + 2];
    block_boxes[threadIdx.x * 4 + 3] = boxes[index * 4 + 3];

  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const Dtype *cur_box = boxes + indexes[cur_box_idx] * 4;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (dev_IoU(cur_box, block_boxes + i * 4) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(boxes_num, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}


template<typename Dtype>
int nms_gpu(const int& boxes_num, int* indexes, const Dtype* scores, const Dtype* proposals ,const Dtype& threshold) {
  if (boxes_num <=1) return boxes_num;
  unsigned long long *mask_dev = NULL;

  const int col_blocks = DIVUP(boxes_num, threadsPerBlock);

  CUDA_CHECK(hipMalloc(&mask_dev,
                        boxes_num * col_blocks * sizeof(unsigned long long)));

  dim3 blocks(DIVUP(boxes_num, threadsPerBlock),
              DIVUP(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);

  nms_kernel < Dtype ><<<blocks, threads>>>(boxes_num, indexes,
          proposals, mask_dev,
          threshold);
  CUDA_POST_KERNEL_CHECK;

  std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
  CUDA_CHECK(hipMemcpy(&mask_host[0],
                        mask_dev,
                        sizeof(unsigned long long) * boxes_num * col_blocks,
                        hipMemcpyDeviceToHost));

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  std::vector<int> keep_out(boxes_num, 0);
  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[i] =  1;
      ++num_to_keep;
      unsigned long long *p = &mask_host[0] + i * col_blocks;

      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }

  {
    thrust::device_ptr<int> dev_ptr = thrust::device_pointer_cast(indexes);
    thrust::device_vector<int> stencil(keep_out.begin(), keep_out.end());
    thrust::remove_if(dev_ptr,dev_ptr+boxes_num,stencil.begin(), thrust_is_zero());
  }

  CUDA_CHECK(hipFree(mask_dev));
  return num_to_keep;
}
template int nms_gpu<double>(const int& boxes_num, int* indexes, const double* scores, const double* proposals ,const double& threshold);

template int nms_gpu<float>(const int& boxes_num, int* indexes, const float* scores, const float* proposals ,const float& threshold);

}//caffe