
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_conv_layer.hpp"

namespace caffe {

//__global__ void sync_conv_groups() { }

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();

#if CUDNN_VERSION_MIN(7,0,0)
    CUDNN_CHECK(cudnnConvolutionForward(Caffe::cudnn_handle(),
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data,
            filter_desc_, weight,
            conv_descs_[i],
            fwd_algo_[i], workspace[0], workspace_fwd_sizes_[i],
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data));
    // Bias.
    if (this->bias_term_) {
      const Dtype* bias_data = this->blobs_[1]->gpu_data();
      CUDNN_CHECK(cudnnAddTensor(Caffe::cudnn_handle(),
            cudnn::dataType<Dtype>::one,
            bias_desc_, bias_data,
            cudnn::dataType<Dtype>::one,
            top_descs_[i], top_data));
    }
#else
    cudaEventRecord(start_event_, Caffe::cuda_stream());

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      cudaStream_t stream;
      cudnnGetStream(handle_[g], &stream);
      cudaStreamWaitEvent(stream,start_event_,0);
      // Filters.
      CUDNN_CHECK(cudnnConvolutionForward(handle_[g],
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            filter_desc_, weight + this->weight_offset_ * g,
            conv_descs_[i],
            fwd_algo_[i], workspace[g], workspace_fwd_sizes_[i],
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data + top_offset_ * g));

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
        CUDNN_CHECK(cudnnAddTensor(handle_[g],
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
      }
      cudaEventRecord(end_event_[g], stream);
      cudaStreamWaitEvent(Caffe::cuda_stream(),end_event_[g],0);
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    //BAD HABIT
    //sync_conv_groups<<<1, 1>>>();
#endif
  }
}

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();

#if CUDNN_VERSION_MIN(7,0,0)
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      CUDNN_CHECK(cudnnConvolutionBackwardBias(Caffe::cudnn_handle(),
            cudnn::dataType<Dtype>::one,
            top_descs_[i],  top_diff,
            cudnn::dataType<Dtype>::one,
            bias_desc_, bias_diff));
    }

    // Gradient w.r.t. weights.
    if (this->param_propagate_down_[0]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      CUDNN_CHECK(cudnnConvolutionBackwardFilter(
            Caffe::cudnn_handle(),
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data,
            top_descs_[i],    top_diff,
            conv_descs_[i],
            bwd_filter_algo_[i], workspace[0],
            workspace_bwd_filter_sizes_[i],
            cudnn::dataType<Dtype>::one,
            filter_desc_, weight_diff ));
    }

    // Gradient w.r.t. bottom data.
    if (propagate_down[i]) {
      if (weight == NULL) {
        weight = this->blobs_[0]->gpu_data();
      }
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      CUDNN_CHECK(cudnnConvolutionBackwardData(
            Caffe::cudnn_handle(),
            cudnn::dataType<Dtype>::one,
            filter_desc_, weight,
            top_descs_[i], top_diff,
            conv_descs_[i],
            bwd_data_algo_[i], workspace[0],
            workspace_bwd_data_sizes_[i],
            cudnn::dataType<Dtype>::zero,
            bottom_descs_[i], bottom_diff));
    }
#else
    cudaEventRecord(start_event_,Caffe::cuda_stream());
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        const int& handle_id = 0*this->group_ + g;
        cudaStream_t stream;
        cudnnGetStream(handle_[handle_id], &stream);
        cudaStreamWaitEvent(stream,start_event_,0);
        CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[handle_id],
              cudnn::dataType<Dtype>::one,
              top_descs_[i],  top_diff + top_offset_ * g,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_diff + bias_offset_ * g));
        cudaEventRecord(end_event_[handle_id],stream);
        cudaStreamWaitEvent(Caffe::cuda_stream(),end_event_[handle_id],0);
      }

      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        const int& handle_id = 1*this->group_ + g;
        cudaStream_t stream;
        cudnnGetStream(handle_[handle_id], &stream);
        cudaStreamWaitEvent(stream,start_event_,0);
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(
              handle_[handle_id],
              cudnn::dataType<Dtype>::one,
              bottom_descs_[i], bottom_data + bottom_offset_ * g,
              top_descs_[i],    top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_filter_algo_[i], workspace[handle_id],
              workspace_bwd_filter_sizes_[i],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight_diff + this->weight_offset_ * g));
        cudaEventRecord(end_event_[handle_id],stream);
        cudaStreamWaitEvent(Caffe::cuda_stream(),end_event_[handle_id],0);
      }

      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          weight = this->blobs_[0]->gpu_data();
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        const int& handle_id = 2*this->group_ + g;
        cudaStream_t stream;
        cudnnGetStream(handle_[handle_id], &stream);
        cudaStreamWaitEvent(stream,start_event_,0);
        CUDNN_CHECK(cudnnConvolutionBackwardData(
              handle_[handle_id],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight + this->weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_data_algo_[i], workspace[handle_id],
              workspace_bwd_data_sizes_[i],
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
        cudaEventRecord(end_event_[handle_id],stream);
        cudaStreamWaitEvent(Caffe::cuda_stream(),end_event_[handle_id],0);
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    //BAD HABIT
    //sync_conv_groups<<<1, 1>>>();
#endif
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
