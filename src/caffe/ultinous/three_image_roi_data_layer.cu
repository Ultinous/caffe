#include <vector>

#include "caffe/ultinous/three_image_roi_data_layer.hpp"

namespace caffe {
namespace ultinous {

template <typename Dtype>
void ThreeImageROIDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  BatchWithBoxes* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  
  //std::cout << "---- ImageROIDataLayer<Dtype>::Forward_gpu" << std::endl;
  
  // Reshape to loaded data.
  std::vector<int> top_shape =  batch->data_.shape();

  top_shape[1] *= top_shape[0];
  top_shape[0] = 1;

  top[0]->Reshape(top_shape);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());

  //std::cout << "---- batch->data_.count()" << batch->data_.count() << std::endl;
  
  // Reshape to image info.
  top[1]->ReshapeLike(batch->info_);
  // Copy info.
  caffe_copy(batch->info_.count(), batch->info_.gpu_data(),
      top[1]->mutable_gpu_data());

  //std::cout << "---- batch->info_.count()" << batch->info_.count() << std::endl;

  // Reshape to image info.
  top[2]->ReshapeLike(batch->bboxes_);
  // Copy bbox.
  caffe_copy(batch->bboxes_.count(), batch->bboxes_.gpu_data(),
      top[2]->mutable_gpu_data());

  //std::cout << "---- batch->bboxes_.count()" << batch->bboxes_.count() << std::endl;

  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(ThreeImageROIDataLayer);

}  // namespace ultinous
}  // namespace caffe
